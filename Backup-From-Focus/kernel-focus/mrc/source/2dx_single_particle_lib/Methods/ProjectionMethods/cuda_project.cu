#include "hip/hip_runtime.h"
#include "cuda_project.h"

#include <fstream>
#include <iostream>

#include <omp.h>

typedef unsigned int uint;

__global__ void proj_kernel_sp2dx(float* out, float size, float size_on_two, float3 mxx, float3 mxy, float3 mxz, hipTextureObject_t tex)
{
	uint x = threadIdx.x;
	uint y = blockIdx.x;
	
	float fx = x-size_on_two;
	float fy = y-size_on_two;
	
	float tx, ty, tz;
	
	float sum = 0;
	
	for(float fz = -size_on_two; fz<size_on_two-.1; fz+=1.0)
	{
		tx = fx*mxx.x + fy*mxx.y + fz*mxx.z + size_on_two + 0.5;
		ty = fx*mxy.x + fy*mxy.y + fz*mxy.z + size_on_two + 0.5;
		tz = fx*mxz.x + fy*mxz.y + fz*mxz.z + size_on_two + 0.5;
		sum += tex3D<float>(tex, tx, ty, tz);
	}
	
	out[x+y*(int)size] = sum;
}


void project_on_gpu(const float* const matrix, float* data, const int nx, const int ny, hipTextureObject_t texObj, hipStream_t& stream)
{
	const dim3 blockSize(ny,1,1);
	const dim3 gridSize(nx,1,1);
	
	float3 mxx, mxy, mxz;
	
	mxx.x = matrix[0];
	mxx.y = matrix[4];
	mxx.z = matrix[8];
	
	mxy.x = matrix[1];
	mxy.y = matrix[5];
	mxy.z = matrix[9];
	
	mxz.x = matrix[2];
	mxz.y = matrix[6];
	mxz.z = matrix[10];
	
	proj_kernel_sp2dx<<<blockSize, gridSize, 0, stream>>>(data, static_cast<float>(nx), static_cast<float>(nx)/2, mxx, mxy, mxz, texObj);
	hipDeviceSynchronize();

	hipError_t err = hipGetLastError();		
	if (err != hipSuccess)
	{
		printf("::CUDA-Error: %s\n", hipGetErrorString(err));
	}
}

